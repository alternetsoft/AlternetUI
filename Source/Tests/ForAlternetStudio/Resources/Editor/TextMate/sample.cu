#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
  cuda_hello<<<1,1>>>(); 
  int a,b,c;
  int *dev_c;
  a=3;
  b=4;
  hipMalloc((void**)&dev_c, sizeof(int));
  add<<<1,1>>>(a,b,dev_c);
  hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d + %d is %d\n", a, b, c);
  hipFree(dev_c);
  return 0;
}